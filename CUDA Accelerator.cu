#include "hip/hip_runtime.h"
#include "CUDA Accelerator.h"

#include "hip/hip_runtime.h"
#include ""

using namespace std;


__global__ void TranslateKernel(float *pPosition, float trans_x, float trans_y, float trans_z, int size, int GridScale)
{
	// Current thread position
	int Row = blockIdx.y * TILE_WIDTH + threadIdx.y;
	int Col = blockIdx.x * TILE_WIDTH + threadIdx.x;

	int rowLength = GridScale * TILE_WIDTH;
	unsigned int index = Row*rowLength + Col;

	if (index < size * 3) { // Within available data
		int rest = index % 3; // Determine dimension
		if (rest == 0) pPosition[index] = pPosition[index] + trans_x;
		else if (rest == 1) pPosition[index] = pPosition[index] + trans_y;
		else if (rest == 2) pPosition[index] = pPosition[index] + trans_z;
	}
}

__global__ void RotateKernel(float *pPosition, float *pPosition_ori, float *pNormal, float *pNormal_ori, float *rotate_mat, int size, int GridScale)
{
	int Row = blockIdx.y * TILE_WIDTH + threadIdx.y;
	int Col = blockIdx.x * TILE_WIDTH + threadIdx.x;

	int rowLength = GridScale * TILE_WIDTH;
	unsigned int index = Row*rowLength + Col;

	if (index < size * 4) {
		int rest = index % 4;
		if (rest == 0) {
			pPosition[index] = // Rotate vertices
				pPosition_ori[index] * rotate_mat[0]
				 + pPosition_ori[index + 1] * rotate_mat[1]
				 + pPosition_ori[index + 2] * rotate_mat[2]
				 + pPosition_ori[index + 3] * rotate_mat[3];
			pNormal[index] =   // Rotate normals
				pNormal_ori[index] * rotate_mat[0]
				+ pNormal_ori[index + 1] * rotate_mat[1]
				+ pNormal_ori[index + 2] * rotate_mat[2]
				+ pNormal_ori[index + 3] * rotate_mat[3];
		}
		else if (rest == 1) {
			pPosition[index] = 
				pPosition_ori[index - 1] * rotate_mat[4]
				+ pPosition_ori[index] * rotate_mat[5]
				+ pPosition_ori[index + 1] * rotate_mat[6]
				+ pPosition_ori[index + 2] * rotate_mat[7];
			pNormal[index] = 
				pNormal_ori[index - 1] * rotate_mat[4]
				+ pNormal_ori[index] * rotate_mat[5]
				+ pNormal_ori[index + 1] * rotate_mat[6]
				+ pNormal_ori[index + 2] * rotate_mat[7];
		}
		else if (rest == 2) {
			pPosition[index] = 
				pPosition_ori[index - 2] * rotate_mat[8]
				+ pPosition_ori[index - 1] * rotate_mat[9]
				+ pPosition_ori[index] * rotate_mat[10]
				+ pPosition_ori[index + 1] * rotate_mat[11];
			pNormal[index] = 
				pNormal_ori[index - 2] * rotate_mat[8]
				+ pNormal_ori[index - 1] * rotate_mat[9]
				+ pNormal_ori[index] * rotate_mat[10]
				+ pNormal_ori[index + 1] * rotate_mat[11];
		}
		else if (rest == 3) {
			pPosition[index] = 
				pPosition_ori[index - 3] * rotate_mat[12]
				+ pPosition_ori[index - 2] * rotate_mat[13]
				+ pPosition_ori[index - 1] * rotate_mat[14]
				+ pPosition_ori[index] * rotate_mat[15];
			pNormal[index] =
				pNormal_ori[index - 3] * rotate_mat[12]
				+ pNormal_ori[index - 2] * rotate_mat[13]
				+ pNormal_ori[index - 1] * rotate_mat[14]
				+ pNormal_ori[index] * rotate_mat[15];
		}
	}
}

__global__ void RescaleKernel(float *pPosition, float scale_x, float scale_y, float scale_z, int size, int GridScale)
{
	int Row = blockIdx.y * TILE_WIDTH + threadIdx.y;
	int Col = blockIdx.x * TILE_WIDTH + threadIdx.x;

	int rowLength = GridScale * TILE_WIDTH;
	unsigned int index = Row*rowLength + Col;

	if (index < size * 3) {
		int rest = index % 3;
		if (rest == 0) pPosition[index] = pPosition[index] * scale_x;
		else if (rest == 1) pPosition[index] = pPosition[index] * scale_y;
		else if (rest == 2) pPosition[index] = pPosition[index] * scale_z;
	}
}

__global__ void ShiftUVKernel(float *pPosition, float shift_x, float shift_y, int size, int GridScale)
{
	int Row = blockIdx.y * TILE_WIDTH + threadIdx.y;
	int Col = blockIdx.x * TILE_WIDTH + threadIdx.x;

	int rowLength = GridScale * TILE_WIDTH;
	unsigned int index = Row*rowLength + Col;

	if (index < size * 2) {
		int rest = index % 2;
		if (rest == 0) pPosition[index] = pPosition[index] + shift_x;
		else if (rest == 1) pPosition[index] = pPosition[index] + shift_y;
	}
}

__global__ void RescaleUVKernel(float *pPosition, float scale_x, float scale_y, int size, int GridScale)
{
	int Row = blockIdx.y * TILE_WIDTH + threadIdx.y;
	int Col = blockIdx.x * TILE_WIDTH + threadIdx.x;

	int rowLength = GridScale * TILE_WIDTH;
	unsigned int index = Row*rowLength + Col;

	if (index < size * 2) {
		int rest = index % 2;
		if (rest == 0) pPosition[index] = pPosition[index] * scale_x;
		else if (rest == 1) pPosition[index] = pPosition[index] * scale_y;
	}
}

__global__ void RotateUVKernel(float *pPosition, float* pPosition_ori, float *rotate_mat, int size, int GridScale)
{
	int Row = blockIdx.y * TILE_WIDTH + threadIdx.y;
	int Col = blockIdx.x * TILE_WIDTH + threadIdx.x;

	int rowLength = GridScale * TILE_WIDTH;
	unsigned int index = Row*rowLength + Col;

	if (index < size * 2) {
		int rest = index % 2;
		if (rest == 0) {
			pPosition[index] =
				pPosition_ori[index] * rotate_mat[0]
				+ pPosition_ori[index + 1] * rotate_mat[1];
		}
		else if (rest == 1) {
			pPosition[index] =
				pPosition_ori[index - 1] * rotate_mat[2]
				+ pPosition_ori[index] * rotate_mat[3];
		}
	}
}

__global__ void STG_TranslateKernel(float *pPosition_y, float trans_y, int size, int GridScale)
{
	int Row = blockIdx.y * TILE_WIDTH + threadIdx.y;
	int Col = blockIdx.x * TILE_WIDTH + threadIdx.x;

	int rowLength = GridScale * TILE_WIDTH;
	unsigned int index = Row*rowLength + Col;

	if (index < size) {
		pPosition_y[index] = pPosition_y[index] + trans_y;
	}
}


extern "C" void translate_CUDA(vector<Vertex> &vertices, vec4 translation)
{
	hipSetDevice(0); // Select CUDA device

	int verticesSize = vertices.size();
	// Device array
	float *pPosition;
	unsigned int size = verticesSize * 3 * sizeof(float);
	hipMalloc((void**)&pPosition, size);

	// Temporary host transfer arrays
	float *temp_Position;
	temp_Position = new float[verticesSize * 3];
	for (int i = 0; i < verticesSize; i++) {
		for (int j = 0; j < 3; j++) {
			temp_Position[3 * i + j] = vertices[i].position[j];
		}
	}

	// Copy data to device
	hipMemcpy(pPosition, temp_Position, size, hipMemcpyHostToDevice);

	int gridScale = ceil(sqrt(verticesSize * 3 / (TILE_WIDTH*TILE_WIDTH)));
	if (gridScale < 1) gridScale = 1; // If the data is too little
	dim3 dimGrid(gridScale, gridScale); // Dimension of a grid
	dim3 dimBlock(TILE_WIDTH, TILE_WIDTH); // Dimension of a block

	float trans_x = translation.x; float trans_y = translation.y; float trans_z = translation.z;

	// Call kernel function
	TranslateKernel <<< dimGrid, dimBlock >>> (pPosition, trans_x, trans_y, trans_z, verticesSize, gridScale);

	// Copy data to host
	hipMemcpy(temp_Position, pPosition, size, hipMemcpyDeviceToHost);

	// Update real data
	for (int i = 0; i < verticesSize; i++) {
		for (int j = 0; j < 3; j++) {
			vertices[i].position[j] = temp_Position[3 * i + j];
		}
	}

	// Release memory
	hipFree(pPosition);
	delete temp_Position;
}

extern "C" void rotate_CUDA(vector<Vertex> &vertices, mat4 rotation)
{
	hipSetDevice(0);

	int verticesSize = vertices.size();

	float *pPosition;
	unsigned int size = verticesSize * 4 * sizeof(float);
	hipMalloc((void**)&pPosition, size);

	float *pPosition_ori;
	hipMalloc((void**)&pPosition_ori, size);

	float *temp_Position;
	temp_Position = new float[verticesSize * 4];
	for (int i = 0; i < verticesSize; i++) {
		for (int j = 0; j < 4; j++) {
			temp_Position[4 * i + j] = vertices[i].position[j];
		}
	}

	hipMemcpy(pPosition, temp_Position, size, hipMemcpyHostToDevice);

	hipMemcpy(pPosition_ori, temp_Position, size, hipMemcpyHostToDevice);

	// Normals
	float *pNormal;
	hipMalloc((void**)&pNormal, size);

	float *pNormal_ori;
	hipMalloc((void**)&pNormal_ori, size);

	float *temp_Normal;
	temp_Normal = new float[verticesSize * 4];
	for (int i = 0; i < verticesSize; i++) {
		for (int j = 0; j < 4; j++) {
			temp_Normal[4 * i + j] = vertices[i].normal[j];
		}
	}

	hipMemcpy(pNormal, temp_Normal, size, hipMemcpyHostToDevice);

	hipMemcpy(pNormal_ori, temp_Normal, size, hipMemcpyHostToDevice);

	// Transform matrix into array
	float *pMatrix;
	int m_size = 16 * sizeof(float);
	hipMalloc((void**)&pMatrix, m_size);

	float *temp_Matrix;
	temp_Matrix = new float[16];
	for (int i = 0; i < 4; i++) {
		for (int j = 0; j < 4; j++) {
			temp_Matrix[4 * i + j] = rotation[i][j];
		}
	}
	hipMemcpy(pMatrix, temp_Matrix, m_size, hipMemcpyHostToDevice);

	int gridScale = ceil(sqrt(verticesSize * 4 / (TILE_WIDTH*TILE_WIDTH)));
	if (gridScale < 1) gridScale = 1;
	dim3 dimGrid(gridScale, gridScale);
	dim3 dimBlock(TILE_WIDTH, TILE_WIDTH);

	RotateKernel <<< dimGrid, dimBlock >>> (pPosition, pPosition_ori, pNormal, pNormal_ori, pMatrix, verticesSize, gridScale);

	hipMemcpy(temp_Position, pPosition, size, hipMemcpyDeviceToHost);

	for (int i = 0; i < verticesSize; i++) {
		for (int j = 0; j < 4; j++) {
			vertices[i].position[j] = temp_Position[4 * i + j];
		}
	}

	hipMemcpy(temp_Normal, pNormal, size, hipMemcpyDeviceToHost);

	for (int i = 0; i < verticesSize; i++) {
		for (int j = 0; j < 4; j++) {
			vertices[i].normal[j] = temp_Normal[4 * i + j];
		}
	}

	hipFree(pPosition);
	hipFree(pPosition_ori);
	hipFree(pNormal);
	hipFree(pNormal_ori);
	hipFree(pMatrix);
	delete temp_Position;
	delete temp_Normal;
	delete temp_Matrix;
}

extern "C" void rescale_CUDA(vector<Vertex> &vertices, vec4 scale)
{
	hipSetDevice(0);

	int verticesSize = vertices.size();

	float *pPosition;
	unsigned int size = verticesSize * 3 * sizeof(float);
	hipMalloc((void**)&pPosition, size);

	float *temp_Position;
	temp_Position = new float[verticesSize * 3];
	for (int i = 0; i < verticesSize; i++) {
		for (int j = 0; j < 3; j++) {
			temp_Position[3 * i + j] = vertices[i].position[j];
		}
	}

	hipMemcpy(pPosition, temp_Position, size, hipMemcpyHostToDevice);

	int gridScale = ceil(sqrt(verticesSize * 3 / (TILE_WIDTH*TILE_WIDTH)));
	if (gridScale < 1) gridScale = 1;
	dim3 dimGrid(gridScale, gridScale);
	dim3 dimBlock(TILE_WIDTH, TILE_WIDTH);

	float scale_x = scale.x; float scale_y = scale.y; float scale_z = scale.z;

	RescaleKernel <<< dimGrid, dimBlock >>> (pPosition, scale_x, scale_y, scale_z, verticesSize, gridScale);

	hipMemcpy(temp_Position, pPosition, size, hipMemcpyDeviceToHost);

	for (int i = 0; i < verticesSize; i++) {
		for (int j = 0; j < 3; j++) {
			vertices[i].position[j] = temp_Position[3 * i + j];
		}
	}

	hipFree(pPosition);
	delete temp_Position;
}

extern "C" void shiftUV_CUDA(vector<Vertex> &vertices, vec4 shift)
{
	hipSetDevice(0);

	int verticesSize = vertices.size();

	float *pPosition;
	unsigned int size = verticesSize * 2 * sizeof(float);
	hipMalloc((void**)&pPosition, size);

	float *temp_Position;
	temp_Position = new float[verticesSize * 2];
	for (int i = 0; i < verticesSize; i++) {
		for (int j = 0; j < 2; j++) {
			temp_Position[2 * i + j] = vertices[i].tex_coords[j];
		}
	}

	hipMemcpy(pPosition, temp_Position, size, hipMemcpyHostToDevice);

	int gridScale = ceil(sqrt(verticesSize * 2 / (TILE_WIDTH*TILE_WIDTH)));
	if (gridScale < 1) gridScale = 1;
	dim3 dimGrid(gridScale, gridScale);
	dim3 dimBlock(TILE_WIDTH, TILE_WIDTH);

	float shift_x = shift.x; float shift_y = shift.y;

	ShiftUVKernel <<< dimGrid, dimBlock >>> (pPosition, shift_x, shift_y, verticesSize, gridScale);

	hipMemcpy(temp_Position, pPosition, size, hipMemcpyDeviceToHost);

	for (int i = 0; i < verticesSize; i++) {
		for (int j = 0; j < 2; j++) {
			vertices[i].tex_coords[j] = temp_Position[2 * i + j];
		}
	}

	hipFree(pPosition);
	delete temp_Position;
}

extern "C" void rescaleUV_CUDA(vector<Vertex> &vertices, vec4 scale)
{
	hipSetDevice(0);

	int verticesSize = vertices.size();

	float *pPosition;
	unsigned int size = verticesSize * 2 * sizeof(float);
	hipMalloc((void**)&pPosition, size);

	float *temp_Position;
	temp_Position = new float[verticesSize * 2];
	for (int i = 0; i < verticesSize; i++) {
		for (int j = 0; j < 2; j++) {
			temp_Position[2 * i + j] = vertices[i].tex_coords[j];
		}
	}

	hipMemcpy(pPosition, temp_Position, size, hipMemcpyHostToDevice);

	int gridScale = ceil(sqrt(verticesSize * 2 / (TILE_WIDTH*TILE_WIDTH)));
	if (gridScale < 1) gridScale = 1;
	dim3 dimGrid(gridScale, gridScale);
	dim3 dimBlock(TILE_WIDTH, TILE_WIDTH);

	float scale_x = scale.x; float scale_y = scale.y;

	RescaleUVKernel <<< dimGrid, dimBlock >>> (pPosition, scale_x, scale_y, verticesSize, gridScale);

	hipMemcpy(temp_Position, pPosition, size, hipMemcpyDeviceToHost);

	for (int i = 0; i < verticesSize; i++) {
		for (int j = 0; j < 2; j++) {
			vertices[i].tex_coords[j] = temp_Position[2 * i + j];
		}
	}

	hipFree(pPosition);
	delete temp_Position;
}

extern "C" void rotateUV_CUDA(vector<Vertex> &vertices, float angle)
{
	hipSetDevice(0);

	int verticesSize = vertices.size();

	float *pPosition;
	unsigned int size = verticesSize * 2 * sizeof(float);
	hipMalloc((void**)&pPosition, size);

	float *pPosition_ori;
	hipMalloc((void**)&pPosition_ori, size);

	float *temp_Position;
	temp_Position = new float[verticesSize * 2];
	for (int i = 0; i < verticesSize; i++) {
		for (int j = 0; j < 2; j++) {
			temp_Position[2 * i + j] = vertices[i].tex_coords[j];
		}
	}

	hipMemcpy(pPosition, temp_Position, size, hipMemcpyHostToDevice);

	hipMemcpy(pPosition_ori, temp_Position, size, hipMemcpyHostToDevice);

	float *temp_Matrix;
	temp_Matrix = new float[4];
	float c = cos(angle), s = sin(angle);
	temp_Matrix[0] = c; temp_Matrix[1] = s;
	temp_Matrix[2] = -s; temp_Matrix[3] = c;

	float *pMatrix;
	int m_size = 4 * sizeof(float);
	hipMalloc((void**)&pMatrix, m_size);

	hipMemcpy(pMatrix, temp_Matrix, m_size, hipMemcpyHostToDevice);

	int gridScale = ceil(sqrt(verticesSize * 2 / (TILE_WIDTH*TILE_WIDTH)));
	if (gridScale < 1) gridScale = 1;
	dim3 dimGrid(gridScale, gridScale);
	dim3 dimBlock(TILE_WIDTH, TILE_WIDTH);

	RotateUVKernel <<< dimGrid, dimBlock >>> (pPosition, pPosition_ori, pMatrix, verticesSize, gridScale);

	hipMemcpy(temp_Position, pPosition, size, hipMemcpyDeviceToHost);

	for (int i = 0; i < verticesSize; i++) {
		for (int j = 0; j < 2; j++) {
			vertices[i].tex_coords[j] = temp_Position[2 * i + j];
		}
	}

	hipFree(pPosition);
	hipFree(pMatrix);
	hipFree(pPosition_ori);
	delete temp_Position;
	delete temp_Matrix;
}

extern "C" void stg_translate_CUDA(vector<vec4> &positions, float compensation)
{
	hipSetDevice(0);

	int verticesSize = positions.size();

	float *pPosition_y;
	unsigned int size = verticesSize * sizeof(float);
	hipMalloc((void**)&pPosition_y, size);

	float *temp_Position_y;
	temp_Position_y = new float[verticesSize];
	for (int i = 0; i < verticesSize; i++) {
		temp_Position_y[i] = positions[i].y;
	}

	hipMemcpy(pPosition_y, temp_Position_y, size, hipMemcpyHostToDevice);

	int gridScale = ceil(sqrt(verticesSize / (TILE_WIDTH*TILE_WIDTH)));
	if (gridScale < 1) gridScale = 1;
	dim3 dimGrid(gridScale, gridScale);
	dim3 dimBlock(TILE_WIDTH, TILE_WIDTH);

	STG_TranslateKernel <<< dimGrid, dimBlock >>> (pPosition_y, compensation, verticesSize, gridScale);

	hipMemcpy(temp_Position_y, pPosition_y, size, hipMemcpyDeviceToHost);

	for (int i = 0; i < verticesSize; i++) {
		positions[i].y = temp_Position_y[i];
	}

	hipFree(pPosition_y);
	delete temp_Position_y;
}